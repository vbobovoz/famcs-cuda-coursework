
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cmath>
#include <iomanip>
#include <omp.h>

#define DIM 16

using namespace std;


// --------------------------------- NORM-CALCULATION ---------------------------------
void multiplyMatrixVector(double* A, double* x, double* result, int N) {
    for(int i = 0; i < N; ++i) {
        result[i] = 0;
        for(int j = 0; j < N; ++j) {
            result[i] += A[i * N + j] * x[j];
        }
    }
}

double computeNorm(double* Ax, double* b, int N) {
    double norm = 0;
    for(int i = 0; i < N; ++i) {
        norm += (Ax[i] - b[i]) * (Ax[i] - b[i]);
    }
    return sqrt(norm);
}

void multiplyMatrixVectorOMP(double* A, double* x, double* result, int N) {
    #pragma omp parallel for
    for(int i = 0; i < N; ++i) {
        result[i] = 0;
        for(int j = 0; j < N; ++j) {
            result[i] += A[i * N + j] * x[j];
        }
    }
}

double computeNormOMP(double* Ax, double* b, int N) {
    double norm = 0;

    #pragma omp parallel for reduction(+:norm)
    for(int i = 0; i < N; ++i) {
        norm += (Ax[i] - b[i]) * (Ax[i] - b[i]);
    }
    return sqrt(norm);
}
// ------------------------------------------------------------------------------------

// ----------------------------------- CPU-NO-TILING -----------------------------------
// Метод Якоби для решения СЛАУ Ax = f
void CPU_NO_TILING_Jacobi_Method(double* A, double* f, double* x, int N, double eps) {
    double* x_prev = new double[N]();
    double* Ax = new double[N]();
    int iterations = 0;

    while(true) { // цикл продолжается до достижения критерия остановки
        // Копируем текущее значение x в x_prev перед каждой итерацией
        for(int i = 0; i < N; ++i) {
            x_prev[i] = x[i];
        }

        for(int i = 0; i < N; ++i) {
            double sum = 0;

            for(int j = 0; j < N; ++j) {
                if(j != i) {
                    sum += A[i * N + j] * x_prev[j];
                }
            }
            
            // Получаем новое приближение x
            x[i] = (f[i] - sum) / A[i * N + i];
        }

        iterations++;

        // Вычисляем Ax^(k)
        multiplyMatrixVector(A, x, Ax, N);

        // Проверка условия остановки
        if(computeNorm(Ax, f, N) <= eps) {
            break;
        }
    }

    delete[] x_prev; // освобождаем память
    delete[] Ax; // освобождаем память
    cout << "Iteration count: " << iterations;
}
// -------------------------------------------------------------------------------------

// ------------------------------------- CPU-OMP -------------------------------------
// Метод Якоби + OpenMP
void CPU_Parallel_Jacobi_Method(double* A, double* f, double* x, int N, double eps) {
    double* x_prev = new double[N]();
    double* Ax = new double[N]();
    int iterations = 0;

    while(true) {
        // Копируем текущее значение x в x_prev параллельно
        #pragma omp parallel for
        for(int i = 0; i < N; ++i) {
            x_prev[i] = x[i];
        }

        // Новое приближение x параллельно
        #pragma omp parallel for
        for(int i = 0; i < N; ++i) {
            double sum = 0;

            // Суммируем элементы в строке, исключая диагональ
            #pragma omp simd
            for(int j = 0; j < N; ++j) {
                if(j != i) {
                    sum += A[i * N + j] * x_prev[j];
                }
            }

            x[i] = (f[i] - sum) / A[i * N + i];
        }

        iterations++;

        // Вычисляем Ax^(k)
        multiplyMatrixVectorOMP(A, x, Ax, N);

        // Проверка условия остановки
        if(computeNormOMP(Ax, f, N) <= eps) {
            break;
        }
    }

    delete[] x_prev; // освобождаем память
    delete[] Ax; // освобождаем память
    cout << "Iteration count: " << iterations;
}
// -----------------------------------------------------------------------------------

// ---------------------------------- GPU-No-Tiling ----------------------------------
__global__ void jacobi_no_tiling_Kernel(double* x_next, const double* A, const double* x_now, const double* b_h, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < N) {
        double sigma = 0.0;
        for(int j = 0; j < N; j++) {
            if(i != j) {
                sigma += A[i * N + j] * x_now[j];
            }
        }
        x_next[i] = (b_h[i] - sigma) / A[i * N + i];
    }
}

void GPU_NO_TILING_Jacobi_Method(double* A, double* f, double* x, int N, double eps) {
    // Размеры блока и сетки
    dim3 DimGrid((N + DIM - 1) / DIM, (N + DIM - 1) / DIM, 1);
    dim3 DimBlock(DIM, DIM, 1);

    // Выделяем память на устройстве
    double* d_A;
    double* d_x_now;
    double* d_x_next;
    double* d_f;
    hipMalloc(&d_A, N * N * sizeof(double));
    hipMalloc(&d_x_now, N * sizeof(double));
    hipMalloc(&d_x_next, N * sizeof(double));
    hipMalloc(&d_f, N * sizeof(double));

    // Копируем данные Host->Device
    hipMemcpy(d_A, A, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x_now, x, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_f, f, N * sizeof(double), hipMemcpyHostToDevice);

    double* x_prev = new double[N];
    double* Ax = new double[N]();
    int iteration = 0;

    while(true) {
        // Копируем текущее приближение в x_prev
        hipMemcpy(x_prev, d_x_now, N * sizeof(double), hipMemcpyDeviceToHost);

        // Запускаем ядро Якоби
        jacobi_no_tiling_Kernel<<<DimGrid, DimBlock>>>(d_x_next, d_A, d_x_now, d_f, N);

        // Копируем результат Device->Host для проверки условия остановки
        hipMemcpy(x, d_x_next, N * sizeof(double), hipMemcpyDeviceToHost);
        
        iteration++;

        // Вычисляем Ax^(k)
        multiplyMatrixVector(A, x, Ax, N);

        // Проверка условия остановки
        if(computeNorm(Ax, f, N) <= eps) {
            break;
        }

        // Обновляем x_now для следующей итерации
        hipMemcpy(d_x_now, d_x_next, N * sizeof(double), hipMemcpyHostToDevice);
    }

    // Вывод числа итераций
    cout << "Iteration count: " << iteration;

    // Освобождаем память
    hipFree(d_A);
    hipFree(d_x_now);
    hipFree(d_x_next);
    hipFree(d_f);
    delete[] x_prev;
    delete[] Ax;
}
// -----------------------------------------------------------------------------------

// ------------------------------------- GENERATION -------------------------------------
// Генерация матрицы с диагональным доминированием
void generateMatrix(double* A, int N) {
    double min = 0;
    double max = 50;
    for(int i = 0; i < N * N; ++i) {
        double tmp = (double)rand() / RAND_MAX; // значение в пределах [0; 1]
        A[i] = min + tmp * (max - min); // присваивание значения в пределах [0; 50]
    }

    for(int i = 0; i < N; ++i) {
        double sum = 0;
        for(int j = 0; j < N; ++j) {
            if(i != j) {
                sum += A[i * N + j];
            }
        }
        // A[i * N + i] += sum + 1; // диагональное доминирование
        A[i * N + i] = sum + 1; // диагональное доминирование
    }
}

// Генерация вектора
void generateVector(double* f, int N) {
    double min = 0;
    double max = 50;
    for(int i = 0; i < N; ++i) {
        double tmp = (double)rand() / RAND_MAX; // значение в пределах [0; 1]
        f[i] = min + tmp * (max - min); // присваивание значения в пределах [0; 50]
    }
}
// --------------------------------------------------------------------------------------

// --------------------------------------- PRINT ---------------------------------------
void printMatrix(double* A, int N, int precision) {
    cout << fixed << setprecision(precision); // устанавливаем точность вывода
    for(int i = 0; i < N; ++i) {
        for(int j = 0; j < N; ++j) {
            cout << A[i * N + j] << " ";
        }
        cout << endl;
    }
    cout << endl;
}

void printVector(double* v, int N, int precision) {
    cout << fixed << setprecision(precision); // устанавливаем точность вывода
    for(int i = 0; i < N; ++i) {
        cout << v[i] << " ";
    }
    cout << endl << endl;
}
// -------------------------------------------------------------------------------------

int main() {
    srand(time(NULL)); // инициализация генератора случайных чисел
 
    int N = 256; // размер матрицы и вектора
    double eps = 1e-5; // погрешность
    int precision = 10; // количество знаков после запятой в выводе

    double* A = new double[N * N]; // матрица A
    double* f = new double[N]; // вектор f
    double* x_cpu = new double[N](); // начальное приближение CPU_NO_TILING (нулевой вектор)
    double* x_cpu_omp = new double[N](); // начальное приближение CPU_OMP (нулевой вектор)
    double* x_gpu = new double[N](); // начальное приближение GPU_NO_TILING (нулевой вектор)

    // События для измерения времени
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Генерация матрицы и вектора
    generateMatrix(A, N);
    generateVector(f, N);

    // cout << "Матрица A:" << endl;
    // printMatrix(A, N, precision);

    // cout << "Вектор f:" << endl;
    // printVector(f, N, precision);

    // Вывод времени
    printf("\nTIME:");

    // -------------------------------- CPU --------------------------------    
    hipEventRecord(start, 0); // старт таймера
    CPU_NO_TILING_Jacobi_Method(A, f, x_cpu, N, eps); // вызов метода Якоби

    float timerValueCPU;
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&timerValueCPU, start, stop);
    printf("\n СPU         %f msec", timerValueCPU);

    // Вывод результата
    // cout << "Решение x_cpu:" << endl;
    // printVector(x_cpu, N, precision);
    // ---------------------------------------------------------------------    

    // -------------------------------- CPU-OMP --------------------------------
    hipEventRecord(start, 0); // старт таймера
    CPU_Parallel_Jacobi_Method(A, f, x_cpu_omp, N, eps); // вызов метода Якоби

    float timerValueCPUOpenMP;
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&timerValueCPUOpenMP, start, stop);
    printf("\n СPU(OpenMP) %f msec", timerValueCPUOpenMP);

    // Вывод результата
    // cout << "Решение x_cpu_omp:" << endl;
    // printVector(x_cpu_omp, N, precision);
    // -------------------------------------------------------------------------

    // -------------------------------- GPU --------------------------------
    hipEventRecord(start, 0); // старт таймера
    GPU_NO_TILING_Jacobi_Method(A, f, x_gpu, N, eps); // вызов метода Якоби

    float timerValueGPU;
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&timerValueGPU, start, stop);
    printf("\n GPU         %f msec", timerValueGPU);

    // Вывод результата
    // cout << "Решение x_gpu:" << endl;
    // printVector(x_gpu, N, precision);
    // ---------------------------------------------------------------------

    // Вывод ускорения
    printf("\nACCELERATION:");
    printf("\n CPU / CPU(OpenMP)  %fx", timerValueCPU / timerValueCPUOpenMP);
    printf("\n CPU / GPU          %fx", timerValueCPU / timerValueGPU);
    printf("\n CPU(OpenMP) / GPU  %fx", timerValueCPUOpenMP / timerValueGPU);

    // Освобождение памяти
    delete[] A;
    delete[] f;
    delete[] x_cpu;
    delete[] x_cpu_omp;
    delete[] x_gpu;

    return 0;
}
